#include "hip/hip_runtime.h"
﻿#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <>

static constexpr size_t NELEMENTS { 1024 * 1024 * 1024 }; // 1 GiB

template<typename scalar_t, typename generator_t> __global__ static void fill_randoms(scalar_t* const device_array, const unsigned size) {
    //
}

auto wmain() -> int {
    //

    ::hipMalloc();

    return EXIT_SUCCESS;
}