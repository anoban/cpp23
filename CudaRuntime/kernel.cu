#include "hip/hip_runtime.h"
﻿#include <algorithm>
#include <cstdio>
#include <numeric>
#include <random>

#include <hip/hip_runtime.h>
#include <>

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x + threadIdx.y + threadIdx.z;
    c[i]  = a[i] + b[i];
}

int main() {
    // std::random_device rdevice {};
    // std::mt19937       rand_eng { rdevice() };
    srand(time(nullptr));
    const int arraySize = 800;
    int       a[arraySize];
    int       b[arraySize];
    int       c[arraySize];

    int64_t sum_dev {}, sum_host {};

    std::generate(std::begin(a), std::end(a), rand);
    std::generate(std::begin(b), std::end(b), rand);

    sum_host               += std::accumulate(std::begin(a), std::end(a), 0i64, std::plus<int> {});
    sum_host               += std::accumulate(std::begin(b), std::end(b), 0i64, std::plus<int> {});

    // Add vectors in parallel.
    hipError_t cudaStatus  = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    for (size_t i = 0; i < arraySize; ++i) {
        // wprintf_s(L"%d + %d = %d\n", a[i], b[i], c[i]);
        sum_dev += c[i];
    }

    wprintf_s(L"host sum = %lld, device sum = %lld\n", sum_host, sum_dev);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size) {
    int*        dev_a = 0;
    int*        dev_b = 0;
    int*        dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**) &dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**) &dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**) &dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
