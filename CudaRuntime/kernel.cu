#include "hip/hip_runtime.h"
﻿// nvcc .\kernel.cu -std=c++20 -O3 -o .\kernel.exe

#include <algorithm>
#include <array>
#include <cstdio>
#include <numeric>
#include <random>
#include <ranges>

#include <hip/hip_runtime.h>
#include <>

template<typename scalar_t> requires std::is_scalar_v<scalar_t>
__global__ void addKernel(_Inout_ scalar_t* out, _In_ const scalar_t* const in_0, _In_ const scalar_t* const in_1) {
    const auto i { threadIdx.x + threadIdx.y + threadIdx.z };
    out[i] = in_0[i] + in_1[i];
    return;
}

static constexpr size_t nthreads { 450 };

template<typename T, typename = std::enable_if<std::is_scalar<T>::value, T>::type> static constexpr size_t memsize = sizeof(T) * nthreads;

// Helper function for using CUDA to add vectors in parallel.
template<typename scalar_t>
hipError_t addWithCuda(
    _Inout_ std::array<scalar_t, nthreads>& out,
    _In_ const std::array<scalar_t, nthreads>& in_0,
    _In_ const std::array<scalar_t, nthreads>& in_1,
    _In_opt_ typename std::enable_if<std::is_scalar<scalar_t>::value, scalar_t>::type = static_cast<scalar_t>(0)
) noexcept {
    scalar_t*   dev_in0 {};
    scalar_t*   dev_in1 {};
    scalar_t*   dev_out {};
    hipError_t cudaStatus {};

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = ::hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        ::fputws(L"hipSetDevice failed!  Do you have a CUDA-capable GPU installed?", stderr);
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = ::hipMalloc(reinterpret_cast<void**>(&dev_out), memsize<scalar_t>);
    if (cudaStatus != hipSuccess) {
        ::fputws(L"hipMalloc failed!", stderr);
        goto Error;
    }

    cudaStatus = ::hipMalloc(reinterpret_cast<void**>(&dev_in0), memsize<scalar_t>);
    if (cudaStatus != hipSuccess) {
        ::fputws(L"hipMalloc failed!", stderr);
        goto Error;
    }

    cudaStatus = ::hipMalloc(reinterpret_cast<void**>(&dev_in1), memsize<scalar_t>);
    if (cudaStatus != hipSuccess) {
        ::fputws(L"hipMalloc failed!", stderr);
        goto Error;
    }

    // copy input arrays from host memory to GPU buffers.
    cudaStatus = ::hipMemcpy(dev_in0, in_0.data(), memsize<scalar_t>, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        ::fputws(L"hipMemcpy failed!", stderr);
        goto Error;
    }

    cudaStatus = ::hipMemcpy(dev_in1, in_1.data(), memsize<scalar_t>, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        ::fputws(L"hipMemcpy failed!", stderr);
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<scalar_t><<<1, nthreads>>>(dev_out, dev_in0, dev_in1);

    // Check for any errors launching the kernel
    cudaStatus = ::hipGetLastError();
    if (cudaStatus != hipSuccess) {
        ::fwprintf_s(stderr, L"addKernel launch failed: %S\n", ::hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = ::hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        ::fwprintf_s(stderr, L"hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = ::hipMemcpy(out.data(), dev_out, memsize<scalar_t>, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        ::fputws(L"hipMemcpy failed!", stderr);
        goto Error;
    }

Error:
    ::hipFree(dev_out);
    ::hipFree(dev_in0);
    ::hipFree(dev_in1);

    return cudaStatus;
}

int wmain() {
    std::array<float, nthreads> a {};
    std::array<float, nthreads> b {};
    std::array<float, nthreads> c {};

    std::random_device rdevice {};
    std::mt19937_64    rand_engine { rdevice() };

    // fill arrays a and b with random floats
    std::generate(a.begin(), a.end(), [&rand_engine]() noexcept {
        return static_cast<float>(rand_engine() / static_cast<double>(RAND_MAX));
    });
    std::generate(b.begin(), b.end(), [&rand_engine]() noexcept {
        return static_cast<float>(rand_engine() / static_cast<double>(RAND_MAX));
    });

    const auto host_sum { std::accumulate(a.cbegin(), a.cend(), 0.0F, std::plus<float> {}) +
                          std::accumulate(b.cbegin(), b.cend(), 0.0F, std::plus<float> {}) };

    ::_putws(L"so far so good :)");

    hipError_t cudaStatus { ::addWithCuda<float>(c, a, b) };
    if (cudaStatus != hipSuccess) {
        ::fputws(L"addWithCuda failed!", stderr);
        return EXIT_FAILURE;
    }

    ::_putws(L"kernel launch is over :)");

    for (const auto& i : std::ranges::views::iota(0LLU, nthreads)) ::wprintf_s(L"%.4f + %.4f = %.4f\n", a.at(i), b.at(i), c.at(i));
    // for (size_t i {}; i < nthreads; ++i) ::wprintf_s(L"%.4f + %.4f = %.4f\n", a.at(i), b.at(i), c.at(i));

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = ::hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        ::fputws(L"hipDeviceReset failed!", stderr);
        return EXIT_FAILURE;
    }

    const auto device_sum { std::reduce(c.cbegin(), c.cend(), 0.0F, std::plus<float> {}) };

    ::_putws(L"all's good :)");
    ::wprintf_s(L"host :: %.5f, device :: %.5f\n", host_sum, device_sum);

    return EXIT_SUCCESS;
}
