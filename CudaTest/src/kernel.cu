﻿#include <parser.cuh>

static constexpr size_t N_RECORDS { 13'612 };

template<std::floating_point T> [[nodiscard]] static constexpr record<T> parse_line(_In_ const std::string_view& line) noexcept {
    // a typical row will be in the format of,
    // 28395,610.291,208.178116708527,173.888747041636,1.19719142411602,0.549812187138347,28715,190.141097274511,0.763922518159806,0.988855998607,0.958027126250128,0.913357754795763,0.00733150613518321,0.00314728916733569,0.834222388245556,0.998723889013168,SEKER
    record<T>         temporary {};
    const char* const cstart { line.data() };
    const char*       begin { line.data() };

    size_t caret = line.find(',', 0); // the first comma
    std::from_chars(begin, cstart + caret /* this delimiter is exclusive */, temporary.area);
    begin = cstart + caret + 1; // char next to the first comma

    caret = line.find(',', caret + 1); // the second comma
    std::from_chars(begin, /* char next to the comma */ cstart + caret, temporary.perimeter);
    begin = cstart + caret + 1;

    caret = line.find(',', caret + 1);
    std::from_chars(begin, cstart + caret, temporary.major_axis_length);
    begin = cstart + caret + 1;

    caret = line.find(',', caret + 1);
    std::from_chars(begin, cstart + caret, temporary.minor_axis_length);
    begin = cstart + caret + 1;

    caret = line.find(',', caret + 1);
    std::from_chars(begin, cstart + caret, temporary.aspect_ratio);
    begin = cstart + caret + 1;

    caret = line.find(',', caret + 1);
    std::from_chars(begin, cstart + caret, temporary.eccentricity);
    begin = cstart + caret + 1;

    caret = line.find(',', caret + 1);
    std::from_chars(begin, cstart + caret, temporary.convex_area);
    begin = cstart + caret + 1;

    caret = line.find(',', caret + 1);
    std::from_chars(begin, cstart + caret, temporary.equiv_diameter);
    begin = cstart + caret + 1;

    caret = line.find(',', caret + 1);
    std::from_chars(begin, cstart + caret, temporary.extent);
    begin = cstart + caret + 1;

    caret = line.find(',', caret + 1);
    std::from_chars(begin, cstart + caret, temporary.roundness);
    begin = cstart + caret + 1;

    caret = line.find(',', caret + 1);
    std::from_chars(begin, cstart + caret, temporary.compactness);
    begin = cstart + caret + 1;

    caret = line.find(',', caret + 1);
    std::from_chars(begin, cstart + caret, temporary.extent);
    begin = cstart + caret + 1;

    caret = line.find(',', caret + 1);
    std::from_chars(begin, cstart + caret, temporary.shape_factor_1);
    begin = cstart + caret + 1;

    caret = line.find(',', caret + 1);
    std::from_chars(begin, cstart + caret, temporary.shape_factor_2);
    begin = cstart + caret + 1;

    caret = line.find(',', caret + 1);
    std::from_chars(begin, cstart + caret, temporary.shape_factor_3);
    begin = cstart + caret + 1;

    caret = line.find(',', caret + 1);
    std::from_chars(begin, cstart + caret, temporary.shape_factor_4);
    begin = cstart + caret + 1;

    // handle the string literal @ the end

    return temporary;
}

template<typename T>
[[nodiscard]] static std::enable_if<std::is_floating_point<T>::value, std::vector<::record<T>>>::type parse_beans_csv(
    _In_ const std::string& csv, _In_ const bool& has_header
) noexcept {
    const auto nlines { std::ranges::count(csv, '\n') }; // 13,612
    assert(nlines == N_RECORDS);

    std::vector<::record<T>> records {};
    records.reserve(nlines); // space for 1 extra record is there since we will not parse the header

    size_t line_begin { has_header ? csv.find('\n', 0) : 0 };
    size_t line_end {};

    while ((line_end = csv.find('\n', line_begin + 1)) != std::string::npos) {
        // create a temporary, given delimiters
        records.push_back(::parse_line<T>(std::string_view { csv.data() + line_begin, csv.data() + line_end }));
        line_begin = line_end;
    }

    return records;
}

auto main() -> int {
    static char current_working_directory[MAX_PATH] {};
    ::GetCurrentDirectoryA(MAX_PATH, current_working_directory);
    ::puts(current_working_directory);

    unsigned long fsize {};
    std::string   beans { ::open(LR"(dry_beans.csv)", &fsize) };

    const auto rows { ::parse_beans_csv<float>(beans, true) };

    std::cout << std::setprecision(15);
    for (const auto& row : rows) std::cout << row;

    return EXIT_SUCCESS;
}
